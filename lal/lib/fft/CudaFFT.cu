#include <lal/LALDatatypes.h>
#include "CudaFunctions.h"

void XLALCudaError(hipError_t error, const char *file, int line)
{
    if(error != hipSuccess)
    {
        fprintf( stderr, "%s:%d %s\n", file, line, hipGetErrorString(error));
        exit(1);
    }
}

void XLALCudaFFTError(hipfftResult_t error, const char *file, int line)
{
    if(error != HIPFFT_SUCCESS)
    {
	/* As there are no GetErrorString function available for CUDA FFT,
	 * the error messages had to be hard-coded,
	 * and needs to be updated with new CUDA releases.
	 */
	switch( error )
	{
	    case HIPFFT_INVALID_PLAN:
	      fprintf( stderr, "%s:%d The plan handle is invalid\n", file, line );
	      break;

	    case HIPFFT_INVALID_VALUE:
	      fprintf( stderr, "%s:%d The input data and/or output data is not valid\n", file, line );
	      break;

	    case HIPFFT_INTERNAL_ERROR:
	      fprintf( stderr, "%s:%d Internal driver error is detected\n", file, line );
	      break;

	    case HIPFFT_EXEC_FAILED:
	      fprintf( stderr, "%s:%d CUFFT failed to execute the transform on GPU\n", file, line );
	      break;

	    case HIPFFT_SETUP_FAILED:
	      fprintf( stderr, "%s:%d CUFFT library failed to initialize\n", file, line );
	      break;

	    default:
		fprintf( stderr, "%s:%d Cuda FFT Error: %d\n", file, line, error);
	}
	exit(1);
    }
}

int cudafft_execute_r2c(hipfftHandle plan,
    hipfftComplex *output, const hipfftReal *input,
    hipfftComplex *d_output, hipfftReal *d_input,UINT4 size)
{
    UINT4 inputBytes = size * sizeof(hipfftReal);
    UINT4 outputBytes = (size/2 + 1) * sizeof(hipfftComplex);

    XLALCUDACHECK(hipMemcpy( d_input, input, inputBytes, hipMemcpyHostToDevice ));

    XLALCUDAFFTCHECK(hipfftExecR2C(plan, d_input, d_output));

    XLALCUDACHECK(hipMemcpy( output, d_output, outputBytes, hipMemcpyDeviceToHost ));

    return 0;
}

int cudafft_execute_c2r(hipfftHandle plan,
    hipfftReal *output, const hipfftComplex *input,
    hipfftReal *d_output, hipfftComplex *d_input, UINT4 size)
{
    UINT4 inputBytes = (size/2 + 1) * sizeof(hipfftComplex);
    UINT4 outputBytes = size * sizeof(hipfftReal);

    XLALCUDACHECK(hipMemcpy( d_input, input, inputBytes, hipMemcpyHostToDevice ));

    XLALCUDAFFTCHECK(hipfftExecC2R(plan, d_input, d_output));

    XLALCUDACHECK(hipMemcpy( output, d_output, outputBytes, hipMemcpyDeviceToHost ));

    return 0;
}

int cudafft_execute_c2c(hipfftHandle plan,
    hipfftComplex *output, const hipfftComplex *input,
    hipfftComplex *d_output, hipfftComplex *d_input,
    INT4 direction, UINT4 size)
{
    UINT4 nBytes = size * sizeof(hipfftComplex);

    XLALCUDACHECK(hipMemcpy( d_input, input, nBytes, hipMemcpyHostToDevice ));

    XLALCUDAFFTCHECK(hipfftExecC2C(plan, d_input, d_output, direction));

    XLALCUDACHECK(hipMemcpy( output, d_output, nBytes, hipMemcpyDeviceToHost ));

    return 0;
}
